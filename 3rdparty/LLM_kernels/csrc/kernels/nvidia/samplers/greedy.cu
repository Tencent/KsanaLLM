#include "hip/hip_runtime.h"
/*
 * Copyright 2024 Tencent Inc.  All rights reserved.
 */

#include "greedy.h"

#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <hipcub/hipcub.hpp>

#include "csrc/utils/nvidia/cuda_utils.h"

using namespace llm_kernels::utils;

namespace llm_kernels {
namespace nvidia {

template <typename T>
__device__ void InvokeWrapArgMax(volatile T* s_max_values, volatile uint32_t* s_argmax) {
  if (static_cast<T>(s_max_values[threadIdx.x]) < static_cast<T>(s_max_values[threadIdx.x + DEFAULT_CUDA_WARP_SIZE])) {
    s_max_values[threadIdx.x] = s_max_values[threadIdx.x + DEFAULT_CUDA_WARP_SIZE];
    s_argmax[threadIdx.x] = s_argmax[threadIdx.x + DEFAULT_CUDA_WARP_SIZE];
  }
  if (static_cast<T>(s_max_values[threadIdx.x]) <
      static_cast<T>(s_max_values[threadIdx.x + DEFAULT_CUDA_HALF_WARP_SIZE])) {
    s_max_values[threadIdx.x] = s_max_values[threadIdx.x + DEFAULT_CUDA_HALF_WARP_SIZE];
    s_argmax[threadIdx.x] = s_argmax[threadIdx.x + DEFAULT_CUDA_HALF_WARP_SIZE];
  }
  if (static_cast<T>(s_max_values[threadIdx.x]) <
      static_cast<T>(s_max_values[threadIdx.x + DEFAULT_CUDA_QUARTER_WARP_SIZE])) {
    s_max_values[threadIdx.x] = s_max_values[threadIdx.x + DEFAULT_CUDA_QUARTER_WARP_SIZE];
    s_argmax[threadIdx.x] = s_argmax[threadIdx.x + DEFAULT_CUDA_QUARTER_WARP_SIZE];
  }
  if (static_cast<T>(s_max_values[threadIdx.x]) <
      static_cast<T>(s_max_values[threadIdx.x + DEFAULT_CUDA_ONE_EIGHTH_WARP_SIZE])) {
    s_max_values[threadIdx.x] = s_max_values[threadIdx.x + DEFAULT_CUDA_ONE_EIGHTH_WARP_SIZE];
    s_argmax[threadIdx.x] = s_argmax[threadIdx.x + DEFAULT_CUDA_ONE_EIGHTH_WARP_SIZE];
  }
  if (static_cast<T>(s_max_values[threadIdx.x]) <
      static_cast<T>(s_max_values[threadIdx.x + DEFAULT_CUDA_ONE_SIXTEENTH_WARP_SIZE])) {
    s_max_values[threadIdx.x] = s_max_values[threadIdx.x + DEFAULT_CUDA_ONE_SIXTEENTH_WARP_SIZE];
    s_argmax[threadIdx.x] = s_argmax[threadIdx.x + DEFAULT_CUDA_ONE_SIXTEENTH_WARP_SIZE];
  }
  if (static_cast<T>(s_max_values[threadIdx.x]) <
      static_cast<T>(s_max_values[threadIdx.x + DEFAULT_CUDA_ONE_THIRTY_TWO_WARP_SIZE])) {
    s_max_values[threadIdx.x] = s_max_values[threadIdx.x + DEFAULT_CUDA_ONE_THIRTY_TWO_WARP_SIZE];
    s_argmax[threadIdx.x] = s_argmax[threadIdx.x + DEFAULT_CUDA_ONE_THIRTY_TWO_WARP_SIZE];
  }
}

template <typename T>
__global__ void InvokeOldArgMaxReduceKernel(const T* input, const int32_t batch_size, const int32_t vocab_size,
                                            uint32_t* result) {
  if (threadIdx.x > vocab_size) {
    return;
  }

  uint32_t border = vocab_size >> 1;

  // NOTE(karlluo): shm consist with DEFAULT_CUDA_BLOCK_THREADS_NUM (float + uint32_t) as following:
  // |-- blockDim.x float --|-- blockDim.x uin32_t --|
  // |     for max value    |    for max index     --|
  // prevent from bank conflict, each thread handle one element `for max value` and `for max index`
  extern __shared__ uint32_t argmax_shm[];
  uint32_t* s_argmax = reinterpret_cast<uint32_t*>(&argmax_shm[blockDim.x]);
  T* s_max_values = reinterpret_cast<T*>(&argmax_shm[0]);

  // NOTE(karlluo): get real value pointer
  uint32_t pos = blockIdx.x;
  T* d_value = const_cast<T*>(input + pos * vocab_size);
  uint32_t* d_index = &(result[blockIdx.x]);

  // NOTE(karlluo): init idx
  uint32_t max_id = threadIdx.x;
  T max_value = d_value[threadIdx.x];

  // NOTE(karlluo): reduce all to shm
  for (uint32_t idx = threadIdx.x; idx < vocab_size; idx += blockDim.x) {
    if (idx < vocab_size && max_value < d_value[idx]) {
      max_id = idx;
      max_value = d_value[idx];
    }
  }

  s_max_values[threadIdx.x] = max_value;
  s_argmax[threadIdx.x] = max_id;

  // NOTE(karlluo): reduce all shm to 32 threads shm
  // get argmax with binary tree
  // each half thread compare the rest half data
  uint32_t compare_idx = max_id;
  for (border = blockDim.x >> 1; border > DEFAULT_CUDA_WARP_SIZE; border >>= 1) {
    if (threadIdx.x > border) {
      return;
    }
    compare_idx = border + threadIdx.x;
    __syncthreads();

    if (compare_idx < blockDim.x && max_value < s_max_values[compare_idx]) {
      max_value = s_max_values[compare_idx];
      max_id = s_argmax[compare_idx];
    }
    s_max_values[threadIdx.x] = max_value;
    s_argmax[threadIdx.x] = max_id;
  }

  // NOTE(karlluo): reduce shm[0, ..., 31] to shm[0]
  if (threadIdx.x < DEFAULT_CUDA_WARP_SIZE) {
    InvokeWrapArgMax(s_max_values, s_argmax);
  }

  if (threadIdx.x == 0) {
    *d_index = static_cast<uint64_t>(s_argmax[0]);
  }
}

template <typename T>
using ArgMaxPair = hipcub::KeyValuePair<int, T>;  // (idx, val)

template <typename T>
__global__ void InvokeArgMaxReduceKernel(const T* input, const int32_t batch_size, const int32_t vocab_size,
                                         uint32_t* result) {
  using BlockReduce = hipcub::BlockReduce<ArgMaxPair<T>, DEFAULT_CUDA_BLOCK_THREADS_NUM>;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  // First reduce in each thread.
  const int offset = blockIdx.x * vocab_size;
  int idx = 0;
  T val = input[offset];
  for (int compare_idx = threadIdx.x; compare_idx < vocab_size; compare_idx += blockDim.x) {
    T compare_val = input[offset + compare_idx];
    if (val < compare_val) {
      idx = compare_idx;
      val = compare_val;
    }
  }

  // Then reduce in the block.
  hipcub::ArgMax argmax_op;
  idx = BlockReduce(temp_storage).Reduce(ArgMaxPair<T>{idx, val}, argmax_op).key;

  // Write result to global memory.
  if (threadIdx.x == 0) {
    result[blockIdx.x] = idx;
  }
}

template <typename T>
void InvokeArgMaxReduce(const T* input, const int32_t batch_size, const int32_t vocab_size, uint32_t* result,
                        hipStream_t& stream) {
  dim3 grid(batch_size);
  dim3 block(DEFAULT_CUDA_BLOCK_THREADS_NUM);

  // By default, the old version of argmax is used, which has issues with multiple maxima.
  // We will transition to the correct new version later.
  if (std::getenv("ENABLE_NEW_ARGMAX") == nullptr) {
    const uint32_t s_mem_size = DEFAULT_CUDA_BLOCK_THREADS_NUM * (sizeof(float) + sizeof(uint32_t));
    InvokeOldArgMaxReduceKernel<<<grid, block, s_mem_size, stream>>>(input, batch_size, vocab_size, result);
  } else {
    InvokeArgMaxReduceKernel<<<grid, block, 0, stream>>>(input, batch_size, vocab_size, result);
  }
}

#define INSTANTIATE_INVOKE_ARG_MAX_REDUCE(T)                                                           \
  template void InvokeArgMaxReduce(const T* input, const int32_t batch_size, const int32_t vocab_size, \
                                   uint32_t* result, hipStream_t& stream);

INSTANTIATE_INVOKE_ARG_MAX_REDUCE(float);
INSTANTIATE_INVOKE_ARG_MAX_REDUCE(half);
INSTANTIATE_INVOKE_ARG_MAX_REDUCE(__hip_bfloat16);

#undef INSTANTIATE_INVOKE_ARG_MAX_REDUCE

}  // namespace nvidia
}  // namespace llm_kernels
