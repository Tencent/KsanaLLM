
#include <hip/hip_runtime.h>
/*
 * Copyright (c) 2025 DeepSeek
 *
 * MIT License
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 *
 * Adapted from
 * [FlashMLA Project] https://github.com/deepseek-ai/FlashMLA/blob/main/csrc/kernels/get_mla_metadata.cu
 */
#ifdef ENABLE_FLASH_MLA

#  include "get_mla_metadata.h"

#  include <cuda_runtime_api.h>
#  include <cutlass/fast_math.h>

#  include "utils.h"

namespace llm_kernels {
namespace nvidia {
__global__ void __launch_bounds__(32, 1, 1)
    get_mla_metadata_kernel(__grid_constant__ const Mla_metadata_params params) {
  int *seqlens_k_ptr = params.seqlens_k_ptr;
  int *tile_scheduler_metadata_ptr = params.tile_scheduler_metadata_ptr;
  int *num_splits_ptr = params.num_splits_ptr;
  int batch_size = params.batch_size;
  int block_size_n = params.block_size_n;
  int fixed_overhead_num_blocks = params.fixed_overhead_num_blocks;
  int num_sm_parts = params.num_sm_parts;

  extern __shared__ int shared_mem[];
  int *num_blocks_shared = shared_mem;               // [batch_size]
  int *num_splits_shared = shared_mem + batch_size;  // [batch_size+1]

  int total_num_blocks = 0;
  for (int i = threadIdx.x; i < batch_size; i += 32) {
    int num_blocks = cutlass::ceil_div(seqlens_k_ptr[i], block_size_n);
    total_num_blocks += num_blocks + fixed_overhead_num_blocks;
    num_blocks_shared[i] = num_blocks;
  }
  for (int offset = 16; offset >= 1; offset /= 2) {
    total_num_blocks += __shfl_xor_sync(uint32_t(-1), total_num_blocks, offset);
  }
  __syncwarp();

  if (threadIdx.x == 0) {
    int payload = max(cutlass::ceil_div(total_num_blocks, num_sm_parts) + fixed_overhead_num_blocks,
                      2 * fixed_overhead_num_blocks);

    int now_idx = 0, now_block = 0, now_n_split_idx = 0, cum_num_splits = 0;
    num_splits_shared[0] = 0;
    for (int i = 0; i < num_sm_parts; ++i) {
      int tile_scheduler_metadata0[4], tile_scheduler_metadata1;
      tile_scheduler_metadata0[0] = now_idx;
      tile_scheduler_metadata0[1] = now_block * block_size_n;
      tile_scheduler_metadata1 = now_n_split_idx;
      int remain_payload = payload;
      while (now_idx < batch_size) {
        int num_blocks = num_blocks_shared[now_idx];
        int now_remain_blocks = num_blocks - now_block;
        if (remain_payload >= now_remain_blocks + fixed_overhead_num_blocks) {
          cum_num_splits += now_n_split_idx + 1;
          num_splits_shared[now_idx + 1] = cum_num_splits;
          remain_payload -= now_remain_blocks + fixed_overhead_num_blocks;
          ++now_idx;
          now_block = 0;
          now_n_split_idx = 0;
        } else {
          if (remain_payload - fixed_overhead_num_blocks > 0) {
            now_block += remain_payload - fixed_overhead_num_blocks;
            ++now_n_split_idx;
            remain_payload = 0;
          }
          break;
        }
      }
      tile_scheduler_metadata0[2] = now_block > 0 ? now_idx : now_idx - 1;
      tile_scheduler_metadata0[3] = now_block > 0 ? now_block * block_size_n : seqlens_k_ptr[now_idx - 1];
      *reinterpret_cast<int4 *>(tile_scheduler_metadata_ptr + i * TileSchedulerMetaDataSize) =
          *reinterpret_cast<int4 *>(tile_scheduler_metadata0);
      tile_scheduler_metadata_ptr[i * TileSchedulerMetaDataSize + 4] = tile_scheduler_metadata1;
    }
    FLASH_DEVICE_ASSERT(now_idx == batch_size && now_block == 0 && now_n_split_idx == 0);
  }
  __syncwarp();

  for (int i = threadIdx.x; i <= batch_size; i += 32) {
    num_splits_ptr[i] = num_splits_shared[i];
  }
}  // namespace nvidia

// Ksana function
void SetMlaMetadataKernelAttribute(const int max_batch_size, cudaStream_t stream) {
  const int max_smem = sizeof(int) * (max_batch_size * 2 + 1);
  CHECK_CUDA(cudaFuncSetAttribute(get_mla_metadata_kernel, cudaFuncAttributeMaxDynamicSharedMemorySize, max_smem));
}

// Ksana function
void GetMlaMetadata(Mla_metadata_params &params, cudaStream_t stream) {
  const int smem_size = sizeof(int) * (params.batch_size * 2 + 1);
  get_mla_metadata_kernel<<<1, 32, smem_size, stream>>>(params);
  CHECK_CUDA_KERNEL_LAUNCH();
}

}  // namespace nvidia
}  // namespace llm_kernels
#endif