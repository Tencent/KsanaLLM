#include "hip/hip_runtime.h"
/*
 * Copyright 2025 Tencent Inc.  All rights reserved.
 */
#include "csrc/kernels/nvidia/per_token_group_quant/per_token_group_quant_8bit.h"

#include <gtest/gtest.h>

#include "csrc/utils/nvidia/cuda_utils.h"
#include "tests/kernels/nvidia/utils/testsuit_base.h"

namespace llm_kernels {
namespace nvidia {
namespace test {

class LlamaNvidiaPerTokenGroupQuantFp8TestSuit : public NvidiaTestSuitBase {
 public:
  void SetUp() override { NvidiaTestSuitBase::SetUp(); }

  void TearDown() override { NvidiaTestSuitBase::TearDown(); }

 protected:
  using NvidiaTestSuitBase::stream;
  int group_size_ = 128;
  const std::vector<std::pair<int, int>> m_n_pairs_ = {{1, 128}};

 protected:
  template <typename T>
  void TestPerTokenGroupQuantFp8(const size_t m, const size_t n, hipStream_t stream, bool is_column_major) {
    size_t data_size = m * n * sizeof(T);
    size_t q_size = m * n;
    size_t s_size = m * n / group_size_;

    // Allocate device memory
    BufferMeta input_meta = CreateBuffer<T>(MemoryType::MEMORY_GPU, {m, n},
                                            /*is_random_init*/ true);
    BufferMeta output_q_meta = CreateBuffer<__hip_fp8_e4m3_fnuz>(MemoryType::MEMORY_GPU, {q_size},
                                                           /*is_random_init*/ false);
    BufferMeta output_s_meta = CreateBuffer<float>(MemoryType::MEMORY_GPU, {s_size},
                                                   /*is_random_init*/ false);
    // Set input data to device
    std::vector<float> h_data_float(m * n);
    for (size_t i = 0; i < m * n; ++i) {
      h_data_float[i] = i / 100.0f;
    }
    if (std::is_same<T, float>::value) {
      CHECK_NVIDIA_CUDA_ERROR(hipMemcpy(input_meta.data_ptr, h_data_float.data(), data_size, hipMemcpyHostToDevice));
    } else if (std::is_same<T, half>::value) {
      std::vector<half> h_data_half(m * n);
      for (size_t i = 0; i < m * n; ++i) {
        h_data_half[i] = __float2half(h_data_float[i]);
      }
      CHECK_NVIDIA_CUDA_ERROR(hipMemcpy(input_meta.data_ptr, h_data_half.data(), data_size, hipMemcpyHostToDevice));
    } else if (std::is_same<T, __hip_bfloat16>::value) {
      std::vector<__hip_bfloat16> h_data_bf16(m * n);
      for (size_t i = 0; i < m * n; ++i) {
        h_data_bf16[i] = __float2bfloat16(h_data_float[i]);
      }
      CHECK_NVIDIA_CUDA_ERROR(hipMemcpy(input_meta.data_ptr, h_data_bf16.data(), data_size, hipMemcpyHostToDevice));
    }

    // Call the kernel
    per_token_group_quant_fp8<T>(input_meta.data_ptr, output_q_meta.data_ptr, output_s_meta.data_ptr, m, n, group_size_,
                                 is_column_major, stream);
    CHECK_NVIDIA_CUDA_ERROR(hipStreamSynchronize(stream));

    // Copy output data back to host
    std::vector<uint8_t> h_q(q_size);
    std::vector<float> h_s(s_size);
    CHECK_NVIDIA_CUDA_ERROR(
        hipMemcpy(h_q.data(), output_q_meta.data_ptr, q_size * sizeof(__hip_fp8_e4m3_fnuz), hipMemcpyDeviceToHost));
    CHECK_NVIDIA_CUDA_ERROR(
        hipMemcpy(h_s.data(), output_s_meta.data_ptr, s_size * sizeof(float), hipMemcpyDeviceToHost));

    // Reference data
    std::vector<float> target_scales = {0.0028, 0.0057};
    std::vector<int> target_q = {
        0,   70,  78,  83,  86,  89,  91,  92,  94,  96,  97,  98,  99,  99,  100, 101, 102, 103, 104, 104, 105, 105,
        106, 106, 107, 107, 107, 108, 108, 109, 109, 110, 110, 111, 111, 111, 112, 112, 112, 113, 113, 113, 113, 113,
        114, 114, 114, 114, 115, 115, 115, 115, 115, 116, 116, 116, 116, 117, 117, 117, 117, 117, 118, 118, 118, 118,
        119, 119, 119, 119, 119, 119, 120, 120, 120, 120, 120, 120, 121, 121, 121, 121, 121, 121, 121, 121, 121, 122,
        122, 122, 122, 122, 122, 122, 122, 122, 123, 123, 123, 123, 123, 123, 123, 123, 123, 124, 124, 124, 124, 124,
        124, 124, 124, 124, 125, 125, 125, 125, 125, 125, 125, 125, 125, 126, 126, 126, 126, 126, 118, 118, 118, 118,
        118, 119, 119, 119, 119, 119, 119, 119, 119, 119, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120, 120,
        120, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 121, 122, 122,
        122, 122, 122, 122, 122, 122, 122, 122, 122, 122, 122, 122, 122, 122, 122, 122, 123, 123, 123, 123, 123, 123,
        123, 123, 123, 123, 123, 123, 123, 123, 123, 123, 123, 123, 123, 124, 124, 124, 124, 124, 124, 124, 124, 124,
        124, 124, 124, 124, 124, 124, 124, 124, 124, 125, 125, 125, 125, 125, 125, 125, 125, 125, 125, 125, 125, 125,
        125, 125, 125, 125, 125, 126, 126, 126, 126, 126, 126, 126, 126, 126};

    // Check the results
    for (size_t i = 0; i < (m * n) / 128; ++i) {
      EXPECT_NEAR(h_s[i], target_scales[i], 1e-3);
    }
    for (size_t i = 0; i < m * n; ++i) {
      EXPECT_NEAR(static_cast<int>(h_q[i]), target_q[i], 1);
    }

    DeleteBuffer(input_meta);
    DeleteBuffer(output_q_meta);
    DeleteBuffer(output_s_meta);
  }
};

TEST_F(LlamaNvidiaPerTokenGroupQuantFp8TestSuit, HalfPerTokenGroupQuantFp8Test) {
  for (const auto& m_n_pair : m_n_pairs_) {
    TestPerTokenGroupQuantFp8<half>(static_cast<size_t>(m_n_pair.first), static_cast<size_t>(m_n_pair.second), stream,
                                    true);
    TestPerTokenGroupQuantFp8<half>(static_cast<size_t>(m_n_pair.first), static_cast<size_t>(m_n_pair.second), stream,
                                    false);
  }
}

TEST_F(LlamaNvidiaPerTokenGroupQuantFp8TestSuit, FloatPerTokenGroupQuantFp8Test) {
  for (const auto& m_n_pair : m_n_pairs_) {
    TestPerTokenGroupQuantFp8<float>(static_cast<size_t>(m_n_pair.first), static_cast<size_t>(m_n_pair.second), stream,
                                     true);
    TestPerTokenGroupQuantFp8<float>(static_cast<size_t>(m_n_pair.first), static_cast<size_t>(m_n_pair.second), stream,
                                     false);
  }
}

TEST_F(LlamaNvidiaPerTokenGroupQuantFp8TestSuit, BFloat16PerTokenGroupQuantFp8Test) {
  for (const auto& m_n_pair : m_n_pairs_) {
    TestPerTokenGroupQuantFp8<__hip_bfloat16>(static_cast<size_t>(m_n_pair.first), static_cast<size_t>(m_n_pair.second),
                                             stream, true);
    TestPerTokenGroupQuantFp8<__hip_bfloat16>(static_cast<size_t>(m_n_pair.first), static_cast<size_t>(m_n_pair.second),
                                             stream, false);
  }
}

}  // namespace test
}  // namespace nvidia
}  // namespace llm_kernels