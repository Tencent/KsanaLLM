#include "hip/hip_runtime.h"
/*
 * Copyright 2025 vLLM Team
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 * Adapted from
 * [vLLM Project]
 * https://github.com/vllm-project/vllm/tree/65334ef3b9e4fd32ebc5c4e512debc25d5025488/csrc/quantization/gptq_marlin
 */

#include "csrc/kernels/nvidia/gptq_marlin/marlin.cuh"
#include "csrc/kernels/nvidia/gptq_marlin/marlin_wrapper.h"
#include "csrc/utils/nvidia/cuda_utils.h"
#include "csrc/utils/nvidia/string_utils.h"

using namespace llm_kernels::utils;

namespace llm_kernels {
namespace nvidia {
namespace marlin {

template <int const num_threads, int const num_bits, bool const has_perm>
__global__ void gptq_marlin_repack_kernel(uint32_t const* __restrict__ b_q_weight_ptr,
                                          uint32_t const* __restrict__ perm_ptr, uint32_t* __restrict__ out_ptr,
                                          int num_experts, int size_k, int size_n) {
  constexpr int pack_factor = 32 / num_bits;

  int k_tiles = size_k / tile_k_size;
  int n_tiles = size_n / tile_n_size;
  int block_k_tiles = div_ceil(k_tiles, gridDim.x);

  auto start_k_tile = blockIdx.x * block_k_tiles;
  if (start_k_tile >= k_tiles) {
    return;
  }

  int expert_idx = blockIdx.y;
  size_t offset = expert_idx * size_k * size_n / pack_factor;
  uint32_t const* expert_b_q_weight_ptr = b_q_weight_ptr + offset;
  uint32_t* expert_out_ptr = out_ptr + offset;
  uint32_t const* expert_perm_ptr = nullptr;
  if constexpr (has_perm) {
    expert_perm_ptr = perm_ptr + expert_idx * size_k;
  }
  int finish_k_tile = min(start_k_tile + block_k_tiles, k_tiles);

  // Wait until the next thread tile has been loaded to shared memory.
  auto wait_for_stage = [&]() {
    // We only have `stages - 2` active fetches since we are double buffering
    // and can only issue the next fetch when it is guaranteed that the previous
    // shared memory load is fully complete (as it may otherwise be
    // overwritten).
    cp_async_wait<repack_stages - 2>();
    __syncthreads();
  };

  extern __shared__ int4 sh[];

  constexpr int perm_size = tile_k_size / 4;

  int4* sh_perm_ptr = sh;
  int4* sh_pipe_ptr = sh_perm_ptr;
  if constexpr (has_perm) {
    sh_pipe_ptr += perm_size;
  }

  constexpr int tile_ints = tile_k_size / pack_factor;

  constexpr int stage_n_threads = tile_n_size / 4;
  constexpr int stage_k_threads = has_perm ? tile_k_size : tile_ints;
  constexpr int stage_size = stage_k_threads * stage_n_threads;

  auto load_perm_to_shared = [&](int k_tile_id) {
    int first_k_int4 = (k_tile_id * tile_k_size) / 4;

    int4 const* perm_int4_ptr = reinterpret_cast<int4 const*>(expert_perm_ptr);

    if (threadIdx.x < perm_size) {
      sh_perm_ptr[threadIdx.x] = perm_int4_ptr[first_k_int4 + threadIdx.x];
    }
    __syncthreads();
  };

  auto fetch_to_shared = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      cp_async_fence();
      return;
    }

    int first_n = n_tile_id * tile_n_size;

    int4* sh_ptr = sh_pipe_ptr + stage_size * pipe;

    if constexpr (has_perm) {
      if (threadIdx.x < stage_size) {
        auto k_id = threadIdx.x / stage_n_threads;
        auto n_id = threadIdx.x % stage_n_threads;

        uint32_t const* sh_perm_int_ptr = reinterpret_cast<uint32_t const*>(sh_perm_ptr);

        int src_k = sh_perm_int_ptr[k_id];
        int src_k_packed = src_k / pack_factor;

        cp_async4(
            &sh_ptr[k_id * stage_n_threads + n_id],
            reinterpret_cast<int4 const*>(&(expert_b_q_weight_ptr[src_k_packed * size_n + first_n + (n_id * 4)])));
      }

    } else {
      if (threadIdx.x < stage_size) {
        auto k_id = threadIdx.x / stage_n_threads;
        auto n_id = threadIdx.x % stage_n_threads;

        int first_k = k_tile_id * tile_k_size;
        int first_k_packed = first_k / pack_factor;

        cp_async4(&sh_ptr[k_id * stage_n_threads + n_id],
                  reinterpret_cast<int4 const*>(
                      &(expert_b_q_weight_ptr[(first_k_packed + k_id) * size_n + first_n + (n_id * 4)])));
      }
    }

    cp_async_fence();
  };

  auto repack_tile = [&](int pipe, int k_tile_id, int n_tile_id) {
    if (n_tile_id >= n_tiles) {
      return;
    }

    auto warp_id = threadIdx.x / 32;
    auto th_id = threadIdx.x % 32;

    if (warp_id >= 4) {
      return;
    }

    int tc_col = th_id / 4;
    int tc_row = (th_id % 4) * 2;

    constexpr int tc_offsets[4] = {0, 1, 8, 9};

    int cur_n = warp_id * 16 + tc_col;

    constexpr int sh_stride = 64;
    constexpr uint32_t mask = (1 << num_bits) - 1;

    int4* sh_stage_ptr = sh_pipe_ptr + stage_size * pipe;
    uint32_t* sh_stage_int_ptr = reinterpret_cast<uint32_t*>(sh_stage_ptr);

    uint32_t* sh_perm_int_ptr = reinterpret_cast<uint32_t*>(sh_perm_ptr);

    uint32_t vals[8];

    if constexpr (has_perm) {
      for (int i = 0; i < 4; i++) {
        int k_idx = tc_row + tc_offsets[i];

        uint32_t src_k = sh_perm_int_ptr[k_idx];
        uint32_t src_k_pos = src_k % pack_factor;

        uint32_t b1_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n];
        uint32_t b1_cur_val = (b1_val >> (src_k_pos * num_bits)) & mask;

        uint32_t b2_val = sh_stage_int_ptr[k_idx * sh_stride + cur_n + 8];
        uint32_t b2_cur_val = (b2_val >> (src_k_pos * num_bits)) & mask;

        vals[i] = b1_cur_val;
        vals[4 + i] = b2_cur_val;
      }

    } else {
      uint32_t b1_vals[tile_ints];
      uint32_t b2_vals[tile_ints];

#pragma unroll
      for (int i = 0; i < tile_ints; i++) {
        b1_vals[i] = sh_stage_int_ptr[cur_n + sh_stride * i];
        b2_vals[i] = sh_stage_int_ptr[cur_n + 8 + sh_stride * i];
      }

#pragma unroll
      for (int i = 0; i < 4; i++) {
        int cur_elem = tc_row + tc_offsets[i];
        int cur_int = cur_elem / pack_factor;
        int cur_pos = cur_elem % pack_factor;

        vals[i] = (b1_vals[cur_int] >> (cur_pos * num_bits)) & mask;
        vals[4 + i] = (b2_vals[cur_int] >> (cur_pos * num_bits)) & mask;
      }
    }

    constexpr int tile_size = tile_k_size * tile_n_size / pack_factor;
    int out_offset = (k_tile_id * n_tiles + n_tile_id) * tile_size;

    // Result of:
    // https://github.com/NVIDIA/FasterTransformer/blob/main/src/fastertransformer/cutlass_extensions/include/cutlass_extensions/interleaved_numeric_conversion.h
    if constexpr (num_bits == 4) {
      constexpr int pack_idx[8] = {0, 2, 4, 6, 1, 3, 5, 7};

      uint32_t res = 0;
#pragma unroll
      for (int i = 0; i < 8; i++) {
        res |= vals[pack_idx[i]] << (i * 4);
      }

      expert_out_ptr[out_offset + th_id * 4 + warp_id] = res;

    } else {
      constexpr int pack_idx[4] = {0, 2, 1, 3};

      uint32_t res1 = 0;
      uint32_t res2 = 0;
#pragma unroll
      for (int i = 0; i < 4; i++) {
        res1 |= vals[pack_idx[i]] << (i * 8);
        res2 |= vals[4 + pack_idx[i]] << (i * 8);
      }

      expert_out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 0] = res1;
      expert_out_ptr[out_offset + th_id * 8 + (warp_id * 2) + 1] = res2;
    }
  };

  auto start_pipes = [&](int k_tile_id, int n_tile_id) {
#pragma unroll
    for (int pipe = 0; pipe < repack_stages - 1; pipe++) {
      fetch_to_shared(pipe, k_tile_id, n_tile_id + pipe);
    }

    wait_for_stage();
  };
#pragma unroll
  for (int k_tile_id = start_k_tile; k_tile_id < finish_k_tile; k_tile_id++) {
    int n_tile_id = 0;

    if constexpr (has_perm) {
      load_perm_to_shared(k_tile_id);
    }

    start_pipes(k_tile_id, n_tile_id);

    while (n_tile_id < n_tiles) {
#pragma unroll
      for (int pipe = 0; pipe < repack_stages; pipe++) {
        fetch_to_shared((pipe + repack_stages - 1) % repack_stages, k_tile_id, n_tile_id + pipe + repack_stages - 1);
        repack_tile(pipe, k_tile_id, n_tile_id + pipe);
        wait_for_stage();
      }
      n_tile_id += repack_stages;
    }
  }
}

#define CALL_IF(NUM_BITS, HAS_PERM)                                                                                  \
  else if (num_bits == NUM_BITS && has_perm == HAS_PERM) {                                                           \
    hipFuncSetAttribute(reinterpret_cast<const void*>(marlin::gptq_marlin_repack_kernel<marlin::repack_threads), NUM_BITS, HAS_PERM>,              \
                         hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem);                               \
    marlin::gptq_marlin_repack_kernel<marlin::repack_threads, NUM_BITS, HAS_PERM>                                    \
        <<<blocks, marlin::repack_threads, max_shared_mem, stream>>>(b_q_weight_ptr, perm_ptr, out_ptr, num_experts, \
                                                                     size_k, size_n);                                \
  }

void gptq_marlin_repack(const uint32_t* b_q_weight_ptr, const uint32_t* perm_ptr, uint32_t* out_ptr,
                        int64_t num_experts, int64_t size_k, int64_t size_n, int64_t num_bits, bool has_perm, int rank,
                        hipStream_t stream) {
  int processors = 0, max_shared_mem = 0;
  hipDeviceGetAttribute(&processors, hipDeviceAttributeMultiprocessorCount, rank);
  hipDeviceGetAttribute(&max_shared_mem, hipDeviceAttributeSharedMemPerBlockOptin, rank);
  KLLM_KERNEL_CHECK(max_shared_mem > 0);
  dim3 blocks(processors, num_experts, 1);

  if (false) {
  }
  CALL_IF(4, false)
  CALL_IF(4, true)
  CALL_IF(8, false)
  CALL_IF(8, true)
  else {
    KLLM_KERNEL_CHECK_WITH_INFO(false,
                                fmtstr("Unsupported repack config: num_bits = {}, has_perm = {}", num_bits, has_perm));
  }
}

std::vector<int64_t> gptq_marlin_repack_meta(int64_t size_k, int64_t size_n, int64_t num_bits) {
  int const pack_factor = 32 / num_bits;
  return {size_k / marlin::tile_size, size_n * marlin::tile_size / pack_factor};
}

}  // namespace marlin
}  // namespace nvidia
}  // namespace llm_kernels
