#include "hip/hip_runtime.h"
/*
 * Modified by Neural Magic
 * Copyright (C) Marlin.2024 Elias Frantar
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *         http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/*
 * Adapted from https://github.com/IST-DASLab/marlin
 */

#include <fmt/format.h>

#include "csrc/kernels/nvidia/gptq_marlin/marlin_template.h"
#include "csrc/kernels/nvidia/gptq_marlin/marlin_wrapper.h"
#include "csrc/utils/nvidia/cuda_utils.h"
#include "csrc/utils/nvidia/string_utils.h"

using namespace llm_kernels::utils;

namespace llm_kernels {
namespace nvidia {
namespace marlin {

__global__ void MarlinDefault(MARLIN_KERNEL_PARAMS){};

using MarlinFuncPtr = void (*)(MARLIN_KERNEL_PARAMS);

// For a given "a" of size [M,K] performs a permutation of the K columns based
// on the given "perm" indices.
__global__ void permute_cols_kernel(int4 const* __restrict__ a_int4_ptr, int const* __restrict__ perm_int_ptr,
                                    int4* __restrict__ out_int4_ptr, int size_m, int size_k, int lda, int block_rows) {
  auto start_row = block_rows * blockIdx.x;
  int finish_row = start_row + block_rows;
  if (finish_row > size_m) {
    finish_row = size_m;
  }
  int cur_block_rows = finish_row - start_row;

  int input_row_stride = lda * sizeof(half) / 16;
  int output_row_stride = size_k * sizeof(half) / 16;

  auto permute_row = [&](int row) {
    int iters = size_k / default_threads;
    int rest = size_k % default_threads;

    int input_offset = row * input_row_stride;
    int output_offset = row * output_row_stride;

    half const* a_row_half = reinterpret_cast<half const*>(a_int4_ptr + input_offset);
    half* out_half = reinterpret_cast<half*>(out_int4_ptr + output_offset);

    int base_k = 0;

    for (int i = 0; i < iters; i++) {
      auto cur_k = base_k + threadIdx.x;
      int src_pos = perm_int_ptr[cur_k];

      out_half[cur_k] = a_row_half[src_pos];

      base_k += default_threads;
    }

    if (rest) {
      if (threadIdx.x < rest) {
        auto cur_k = base_k + threadIdx.x;
        int src_pos = perm_int_ptr[cur_k];

        out_half[cur_k] = a_row_half[src_pos];
      }
    }
  };

  for (int i = 0; i < cur_block_rows; i++) {
    int cur_row = start_row + i;
    if (cur_row < size_m) {
      permute_row(cur_row);
    }
  }
}

typedef struct {
  int thread_k;
  int thread_n;
  int num_threads;
} thread_config_t;

thread_config_t small_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {128, 128, 256},
    {64, 128, 128},
    {128, 64, 128}};

thread_config_t large_batch_thread_configs[] = {
    // Ordered by priority

    // thread_k, thread_n, num_threads
    {64, 256, 256},
    {64, 128, 128},
    {128, 64, 128}};

typedef struct {
  int blocks_per_sm;
  thread_config_t tb_cfg;
} exec_config_t;

int get_scales_cache_size(thread_config_t const& th_config, int prob_m, int prob_n, int prob_k, int num_bits,
                          int group_size, bool has_act_order, bool is_k_full) {
  bool cache_scales_chunk = has_act_order && !is_k_full;

  int tb_n = th_config.thread_n;
  int tb_k = th_config.thread_k;

  // Get max scale groups per thread-block
  int tb_groups;
  if (group_size == -1) {
    tb_groups = 1;
  } else if (group_size == 0) {
    tb_groups = div_ceil(tb_k, 32);  // Worst case is 32 group size
  } else {
    tb_groups = div_ceil(tb_k, group_size);
  }

  if (cache_scales_chunk) {
    int load_groups = tb_groups * pipe_stages * 2;  // Chunk size is 2x pipeline over dim K
    load_groups = max(load_groups, 32);             // We load at least 32 scale groups
    return load_groups * tb_n * 2;
  } else {
    int tb_scales = tb_groups * tb_n * 2;

    return tb_scales * pipe_stages;
  }
}

int get_kernel_cache_size(thread_config_t const& th_config, int thread_m_blocks, int prob_m, int prob_n, int prob_k,
                          int num_bits, int group_size, bool has_act_order, bool is_k_full, int has_zp,
                          int is_zp_float) {
  int pack_factor = 32 / num_bits;

  // Get B size
  int tb_k = th_config.thread_k;
  int tb_n = th_config.thread_n;
  int tb_m = thread_m_blocks * 16;
  int sh_a_size = pipe_stages * (tb_m * tb_k) * 2;
  int sh_b_size = pipe_stages * (tb_k * tb_n / pack_factor) * 4;
  int sh_red_size = tb_m * (tb_n + 8);
  int sh_s_size =
      get_scales_cache_size(th_config, prob_m, prob_n, prob_k, num_bits, group_size, has_act_order, is_k_full);
  int sh_g_idx_size = has_act_order && !is_k_full ? pipe_stages * tb_k / 4 : 0;
  int sh_zp_size = 0;
  if (has_zp) {
    if (is_zp_float)
      sh_zp_size = sh_s_size;
    else if (num_bits == 4)
      sh_zp_size = sh_s_size / 4;
    else if (num_bits == 8)
      sh_zp_size = sh_s_size / 2;
  }

  int total_size = max(sh_b_size, sh_red_size) + sh_a_size + sh_s_size + sh_zp_size + sh_g_idx_size;

  return total_size;
}

bool is_valid_config(thread_config_t const& th_config, int thread_m_blocks, int prob_m, int prob_n, int prob_k,
                     int num_bits, int group_size, bool has_act_order, bool is_k_full, int has_zp, int is_zp_float,
                     int max_shared_mem) {
  // Sanity
  if (th_config.thread_k == -1 || th_config.thread_n == -1 || th_config.num_threads == -1) {
    return false;
  }

  // Verify K/N are divisible by thread K/N
  if (prob_k % th_config.thread_k != 0 || prob_n % th_config.thread_n != 0) {
    return false;
  }

  // Verify min for thread K/N
  if (th_config.thread_n < min_thread_n || th_config.thread_k < min_thread_k) {
    return false;
  }

  // num_threads must be at least 128 (= 4 warps)
  if (th_config.num_threads < 128) {
    return false;
  }

  // Check that pipeline fits into cache
  int cache_size = get_kernel_cache_size(th_config, thread_m_blocks, prob_m, prob_n, prob_k, num_bits, group_size,
                                         has_act_order, is_k_full, has_zp, is_zp_float);
  return cache_size <= max_shared_mem;
}

#define _GET_IF(W_TYPE, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS, M_BLOCK_SIZE_8, GROUP_BLOCKS, NUM_THREADS, \
                IS_ZP_FLOAT)                                                                                          \
  else if (q_type == W_TYPE && thread_m_blocks == THREAD_M_BLOCKS && thread_n_blocks == THREAD_N_BLOCKS &&            \
           thread_k_blocks == THREAD_K_BLOCKS && m_block_size_8 == M_BLOCK_SIZE_8 && group_blocks == GROUP_BLOCKS &&  \
           num_threads == NUM_THREADS && is_zp_float == IS_ZP_FLOAT) {                                                \
    kernel = Marlin<scalar_t, W_TYPE.id(), NUM_THREADS, THREAD_M_BLOCKS, THREAD_N_BLOCKS, THREAD_K_BLOCKS,            \
                    M_BLOCK_SIZE_8, pipe_stages, GROUP_BLOCKS, IS_ZP_FLOAT>;                                          \
  }

// COMMON: cases for (group_blocks in [-1, 2, 4, 8] and is_zp_float == false)
//         this is the most common cases
// BIGGROUP: cases for big group size (group_blocks in [-1, 8])
// FZP: cases for float-zero-point (is_zp_float = true)
// ACT: cases for act order case (group_blocks == 0)
// FP4: cases for nvfp4(e2m1) (group_blocks == 1)
#define COMMON_GET_IF_M1(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)       \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, -1, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, 2, NUM_THREADS, false)   \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, 4, NUM_THREADS, false)   \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, 8, NUM_THREADS, false)   \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, 2, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS, false)

#define COMMON_GET_IF_M234(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)     \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, 2, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS, false)  \
                                                                        \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, 2, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS, false)  \
                                                                        \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, 2, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS, false)

#define COMMON_GET_IF(W_TYPE)            \
  COMMON_GET_IF_M1(W_TYPE, 8, 8, 256)    \
  COMMON_GET_IF_M1(W_TYPE, 8, 4, 128)    \
  COMMON_GET_IF_M1(W_TYPE, 4, 8, 128)    \
  COMMON_GET_IF_M234(W_TYPE, 16, 4, 256) \
  COMMON_GET_IF_M234(W_TYPE, 8, 4, 128)  \
  COMMON_GET_IF_M234(W_TYPE, 4, 8, 128)

#define BIGGROUP_GET_IF_M1(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)     \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, -1, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, 8, NUM_THREADS, false)   \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS, false)

#define BIGGROUP_GET_IF_M234(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)   \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS, false)  \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, -1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, 8, NUM_THREADS, false)

#define BIGGROUP_GET_IF(W_TYPE)            \
  BIGGROUP_GET_IF_M1(W_TYPE, 8, 8, 256)    \
  BIGGROUP_GET_IF_M1(W_TYPE, 8, 4, 128)    \
  BIGGROUP_GET_IF_M1(W_TYPE, 4, 8, 128)    \
  BIGGROUP_GET_IF_M234(W_TYPE, 16, 4, 256) \
  BIGGROUP_GET_IF_M234(W_TYPE, 8, 4, 128)  \
  BIGGROUP_GET_IF_M234(W_TYPE, 4, 8, 128)

#define FP4_GET_IF_M1(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)        \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, 1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, 1, NUM_THREADS, false)

#define FP4_GET_IF_M234(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)       \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, 1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, 1, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, 1, NUM_THREADS, false)

#define FP4_GET_IF(W_TYPE)            \
  FP4_GET_IF_M1(W_TYPE, 8, 8, 256)    \
  FP4_GET_IF_M1(W_TYPE, 8, 4, 128)    \
  FP4_GET_IF_M1(W_TYPE, 4, 8, 128)    \
  FP4_GET_IF_M234(W_TYPE, 16, 4, 256) \
  FP4_GET_IF_M234(W_TYPE, 8, 4, 128)  \
  FP4_GET_IF_M234(W_TYPE, 4, 8, 128)

// We currently have 4-bit models only with group_blocks == 4
#define FZP_GET_IF_M1(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)       \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, 4, NUM_THREADS, true) \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS, true)

#define FZP_GET_IF_M234(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)      \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS, true) \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS, true) \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, 4, NUM_THREADS, true)

#define FZP_GET_IF(W_TYPE)            \
  FZP_GET_IF_M1(W_TYPE, 8, 8, 256)    \
  FZP_GET_IF_M1(W_TYPE, 8, 4, 128)    \
  FZP_GET_IF_M1(W_TYPE, 4, 8, 128)    \
  FZP_GET_IF_M234(W_TYPE, 16, 4, 256) \
  FZP_GET_IF_M234(W_TYPE, 8, 4, 128)  \
  FZP_GET_IF_M234(W_TYPE, 4, 8, 128)

// We currently have 4-bit models only with group_blocks == 4
#define ACT_GET_IF_M1(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)        \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, true, 0, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 1, N_BLOCKS, K_BLOCKS, false, 0, NUM_THREADS, false)

#define ACT_GET_IF_M234(W_TYPE, N_BLOCKS, K_BLOCKS, NUM_THREADS)       \
  _GET_IF(W_TYPE, 2, N_BLOCKS, K_BLOCKS, false, 0, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 3, N_BLOCKS, K_BLOCKS, false, 0, NUM_THREADS, false) \
  _GET_IF(W_TYPE, 4, N_BLOCKS, K_BLOCKS, false, 0, NUM_THREADS, false)

#define ACT_GET_IF(W_TYPE)            \
  ACT_GET_IF_M1(W_TYPE, 8, 8, 256)    \
  ACT_GET_IF_M1(W_TYPE, 8, 4, 128)    \
  ACT_GET_IF_M1(W_TYPE, 4, 8, 128)    \
  ACT_GET_IF_M234(W_TYPE, 16, 4, 256) \
  ACT_GET_IF_M234(W_TYPE, 8, 4, 128)  \
  ACT_GET_IF_M234(W_TYPE, 4, 8, 128)

template <typename scalar_t>
MarlinFuncPtr get_marlin_kernel(const llm_kernels::nvidia::vllm_dtype::ScalarType q_type, int thread_m_blocks,
                                int thread_n_blocks, int thread_k_blocks, bool m_block_size_8, bool has_act_order,
                                bool has_zp, int group_blocks, int num_threads, bool is_zp_float) {
  int num_bits = q_type.size_bits();
  auto kernel = MarlinDefault;
  if (false) {
  }

  COMMON_GET_IF(llm_kernels::nvidia::vllm_dtype::kU4)
  COMMON_GET_IF(llm_kernels::nvidia::vllm_dtype::kU4B8)
  COMMON_GET_IF(llm_kernels::nvidia::vllm_dtype::kU8B128)

  FP4_GET_IF(llm_kernels::nvidia::vllm_dtype::kFE2M1f)

  BIGGROUP_GET_IF(llm_kernels::nvidia::vllm_dtype::kFE4M3fn)

  ACT_GET_IF(llm_kernels::nvidia::vllm_dtype::kU4B8)
  ACT_GET_IF(llm_kernels::nvidia::vllm_dtype::kU8B128)

  if (std::is_same<scalar_t, half>::value) {
    if (false) {
    }
    FZP_GET_IF(llm_kernels::nvidia::vllm_dtype::kU4)
  }

  return kernel;
}

template <typename scalar_t>
exec_config_t determine_exec_config(const llm_kernels::nvidia::vllm_dtype::ScalarType& q_type, int prob_m, int prob_n,
                                    int prob_k, int thread_m_blocks, bool m_block_size_8, int num_bits, int group_size,
                                    bool has_act_order, bool is_k_full, bool has_zp, bool is_zp_float,
                                    int max_shared_mem, int sms) {
  exec_config_t exec_cfg = exec_config_t{1, thread_config_t{-1, -1, -1}};
  thread_config_t* thread_configs = thread_m_blocks > 1 ? large_batch_thread_configs : small_batch_thread_configs;
  int thread_configs_size = thread_m_blocks > 1 ? sizeof(large_batch_thread_configs) / sizeof(thread_config_t)
                                                : sizeof(small_batch_thread_configs) / sizeof(thread_config_t);

  for (int i = 0; i < thread_configs_size; i++) {
    thread_config_t th_config = thread_configs[i];

    if (!is_valid_config(th_config, thread_m_blocks, prob_m, prob_n, prob_k, num_bits, group_size, has_act_order,
                         is_k_full, has_zp, is_zp_float, max_shared_mem)) {
      continue;
    }

    int cache_size = get_kernel_cache_size(th_config, thread_m_blocks, prob_m, prob_n, prob_k, num_bits, group_size,
                                           has_act_order, is_k_full, has_zp, is_zp_float);

    int group_blocks = 0;
    if (!has_act_order) {
      group_blocks = group_size == -1 ? -1 : group_size / 16;
    }

    auto kernel = get_marlin_kernel<scalar_t>(q_type, thread_m_blocks, th_config.thread_n / 16, th_config.thread_k / 16,
                                              m_block_size_8, has_act_order, has_zp, group_blocks,
                                              th_config.num_threads, is_zp_float);

    if (kernel == MarlinDefault) continue;

    // int m_tiles = div_ceil(prob_m, thread_m_blocks * 16);
    // int n_tiles = prob_n / th_config.thread_n;
    // int k_tiles = prob_k / th_config.thread_k;

    return {1, th_config};
  }

  return exec_cfg;
}

template <typename scalar_t>
void marlin_mm(const void* A, const void* B, void* C, void* C_tmp, void* s, void* s2, void* zp, void* g_idx, void* perm,
               void* a_tmp, int prob_m, int prob_n, int prob_k, int lda, void* workspace,
               llm_kernels::nvidia::vllm_dtype::ScalarType const& q_type, bool has_act_order, bool is_k_full,
               bool has_zp, int num_groups, int group_size, int dev, hipStream_t stream, int thread_k_init,
               int thread_n_init, int sms, bool use_atomic_add, bool use_fp32_reduce, bool is_zp_float) {
  if (has_zp) {
    KLLM_KERNEL_CHECK_WITH_INFO(
        q_type == llm_kernels::nvidia::vllm_dtype::kU4 || q_type == llm_kernels::nvidia::vllm_dtype::kU8,
        fmt::format("q_type must be u4 or u8 when has_zp = True. Got = {}", q_type.str()));
  } else {
    KLLM_KERNEL_CHECK_WITH_INFO(
        q_type == llm_kernels::nvidia::vllm_dtype::kU4B8 || q_type == llm_kernels::nvidia::vllm_dtype::kU8B128 ||
            q_type == llm_kernels::nvidia::vllm_dtype::kFE4M3fn || q_type == llm_kernels::nvidia::vllm_dtype::kFE2M1f,
        fmt::format("q_type must be uint4b8, uint8b128, float8_e4m3fn or float4_e2m1f when has_zp = False. Got = {}",
                    q_type.str()));
  }

  KLLM_KERNEL_CHECK_WITH_INFO(prob_m > 0 && prob_n > 0 && prob_k > 0,
                              fmt::format("Invalid MNK = [{}, {}, {}]", prob_m, prob_n, prob_k));

  int group_blocks = 0;
  if (has_act_order) {
    if (is_k_full) {
      KLLM_KERNEL_CHECK(group_size != -1);
      group_blocks = group_size / 16;
      KLLM_KERNEL_CHECK_WITH_INFO(
          prob_k % group_blocks == 0,
          fmt::format("prob_k = {} is not divisible by group_blocks = {}", prob_k, group_blocks));
    } else {
      KLLM_KERNEL_CHECK(group_size == 0);
      group_blocks = 0;
    }
  } else {
    if (group_size == -1) {
      group_blocks = -1;
    } else {
      group_blocks = group_size / 16;
      KLLM_KERNEL_CHECK_WITH_INFO(
          prob_k % group_blocks == 0,
          fmt::format("prob_k = {} is not divisible by group_blocks = {}", prob_k, group_blocks));
    }
  }

  int num_bits = q_type.size_bits();
  const int4* A_ptr = (const int4*)A;
  const int4* B_ptr = (const int4*)B;
  int4* C_ptr = (int4*)C;
  int4* C_tmp_ptr = (int4*)C_tmp;
  const int4* s_ptr = (const int4*)s;
  const uint16_t* s2_ptr = (const uint16_t*)s2;
  const int4* zp_ptr = (const int4*)zp;
  const int* g_idx_ptr = (const int*)g_idx;
  const int* perm_ptr = (const int*)perm;
  int4* a_tmp_ptr = (int4*)a_tmp;

  int* locks = (int*)workspace;

  if (has_act_order) {
    // Permute A columns
    int block_rows = div_ceil(prob_m, sms);
    // avoid ">>>" being formatted to "> > >"
    // clang-format off
    permute_cols_kernel<<<sms, default_threads, 0, stream>>>(
        A_ptr, perm_ptr, a_tmp_ptr, prob_m, prob_k, lda, block_rows);
    // clang-format on
    A_ptr = a_tmp_ptr;
    lda = prob_k;

    // If we have a full K, then we can run the non-act-order version of Marlin
    // (since the weight rows are reordered by increasing group ids, and by
    // having a full K, we have full original groups)
    if (is_k_full) has_act_order = false;
  }

  int max_shared_mem = 0;
  hipDeviceGetAttribute(&max_shared_mem, hipDeviceAttributeSharedMemPerBlockOptin, dev);
  KLLM_KERNEL_CHECK(max_shared_mem > 0);

  int max_par = 16;
  if (prob_n <= 4096) max_par = 16 * 8;
  int max_shared_mem_new = max_shared_mem;
  int rest_m = prob_m;
  int max_thread_m_blocks = 4;
  while (rest_m) {
    int par_count = rest_m / (max_thread_m_blocks * 16);
    if (par_count > max_par) par_count = max_par;
    int prob_m_split = par_count > 0 ? (par_count * (max_thread_m_blocks * 16)) : rest_m;

    int thread_k = thread_k_init;
    int thread_n = thread_n_init;

    int thread_m_blocks = min(div_ceil(prob_m_split, 16), max_thread_m_blocks);
    int m_block_size_8 = prob_m_split <= 8;

    // Set thread config
    exec_config_t exec_cfg;
    thread_config_t thread_tfg;
    if (thread_k != -1 && thread_n != -1) {
      thread_tfg = thread_config_t{thread_k, thread_n, default_threads};
      exec_cfg = exec_config_t{1, thread_tfg};
      KLLM_KERNEL_CHECK_WITH_INFO(prob_n % thread_n == 0,
                                  fmt::format("prob_n = {} is not divisible by thread_n = {}", prob_n, thread_n));
      KLLM_KERNEL_CHECK_WITH_INFO(prob_k % thread_k == 0,
                                  fmt::format("prob_k = {} is not divisible by thread_k = {}", prob_k, thread_k));
    } else {
      // Auto config
      exec_cfg = determine_exec_config<scalar_t>(q_type, prob_m_split, prob_n, prob_k, thread_m_blocks, m_block_size_8,
                                                 num_bits, group_size, has_act_order, is_k_full, has_zp, is_zp_float,
                                                 max_shared_mem, sms);
      thread_tfg = exec_cfg.tb_cfg;
      if (thread_tfg.thread_k == -1 && max_thread_m_blocks > 1) {
        max_thread_m_blocks--;
        continue;
      }
    }

    int num_threads = thread_tfg.num_threads;
    thread_k = thread_tfg.thread_k;
    thread_n = thread_tfg.thread_n;
    int blocks = sms * exec_cfg.blocks_per_sm;
    if (exec_cfg.blocks_per_sm > 1) max_shared_mem_new = max_shared_mem / exec_cfg.blocks_per_sm - 1024;

    int thread_k_blocks = thread_k / 16;
    int thread_n_blocks = thread_n / 16;

    KLLM_KERNEL_CHECK_WITH_INFO(
        is_valid_config(thread_tfg, thread_m_blocks, prob_m_split, prob_n, prob_k, num_bits, group_size, has_act_order,
                        is_k_full, has_zp, is_zp_float, max_shared_mem_new),
        fmt::format(
            "Invalid thread config: thread_m_blocks = {}, thread_k = {}, thread_n = {}, num_threads = {} for MKN = "
            "[{}, {}, {}] and num_bits = {}, prob_m_split = {}, group_size = {}, has_act_order = {}, is_k_full = "
            "{}, has_zp = {}, is_zp_float = {}, max_shared_mem_new = {}",
            thread_m_blocks, thread_tfg.thread_k, thread_tfg.thread_n, thread_tfg.num_threads, prob_m, prob_k, prob_n,
            num_bits, prob_m_split, group_size, has_act_order, is_k_full, has_zp, is_zp_float, max_shared_mem_new));

    auto kernel = get_marlin_kernel<scalar_t>(q_type, thread_m_blocks, thread_n_blocks, thread_k_blocks, m_block_size_8,
                                              has_act_order, has_zp, group_blocks, num_threads, is_zp_float);

    if (kernel == MarlinDefault) {
      KLLM_KERNEL_CHECK_WITH_INFO(
          false, fmt::format("Unsupported shapes: MNK = [{}, {}, {}], has_act_order = {}, num_groups = "
                             "{}, group_size = {}, prob_m_split = {}, thread_m_blocks = {}, "
                             "thread_n_blocks = {}, thread_k_blocks = {}, num_threads = {}, num_bits = {}",
                             prob_m, prob_n, prob_k, has_act_order, num_groups, group_size, prob_m_split,
                             thread_m_blocks, thread_n_blocks, thread_k_blocks, num_threads, num_bits));
    }

    hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, max_shared_mem_new);

    bool part_use_atomic_add = use_atomic_add && div_ceil(prob_m_split, 64) * prob_n <= 2048;

    // avoid ">>>" being formatted to "> > >"
    // clang-format off
    kernel<<<blocks, num_threads, max_shared_mem_new, stream>>>(
        A_ptr, B_ptr, C_ptr, C_tmp_ptr, s_ptr, s2_ptr, zp_ptr, g_idx_ptr, num_groups,
        prob_m_split, prob_n, prob_k, lda, locks, part_use_atomic_add,
        use_fp32_reduce, max_shared_mem_new);
    // clang-format on

    A_ptr += prob_m_split * (lda / 8);
    C_ptr += prob_m_split * (prob_n / 8);
    rest_m -= prob_m_split;
  }
}

template <typename T>
void gptq_marlin_gemm(void* a, void* a_tmp, void* b_q_weight, void* b_scales, void* b_zeros, void* g_idx, void* perm,
                      void* workspace, void* c, void* c_tmp, int64_t size_m, int64_t size_n, int64_t size_k,
                      int64_t num_groups, bool is_k_full, bool use_atomic_add, bool use_fp32_reduce, bool is_zp_float,
                      bool has_zp, bool has_act_order, bool is_awq, int rank, hipStream_t stream) {
  if constexpr (std::is_same_v<T, float>) {
    KLLM_KERNEL_THROW("gptq_marlin_gemm not support float type.");
  } else {
    llm_kernels::nvidia::vllm_dtype::ScalarType b_type =
        is_awq ? llm_kernels::nvidia::vllm_dtype::kU4 : llm_kernels::nvidia::vllm_dtype::kU4B8;

    int thread_k = -1;
    int thread_n = -1;
    int sms = -1;
    hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, rank);

    // Detect groupsize and act_order
    int group_size = -1;

    if (has_act_order) {
      if (is_k_full) {
        group_size = size_k / num_groups;
      } else {
        group_size = 0;
      }
    } else {
      if (num_groups > 1) {
        group_size = size_k / num_groups;
      } else {
        group_size = -1;
      }
    }

    marlin::marlin_mm<T>(
        reinterpret_cast<T*>(a), reinterpret_cast<void*>(b_q_weight), reinterpret_cast<T*>(c),
        reinterpret_cast<float*>(c_tmp), reinterpret_cast<T*>(b_scales), nullptr, reinterpret_cast<void*>(b_zeros),
        reinterpret_cast<void*>(g_idx), reinterpret_cast<void*>(perm), reinterpret_cast<T*>(a_tmp), size_m, size_n,
        size_k, size_k, reinterpret_cast<void*>(workspace), b_type, has_act_order, is_k_full, has_zp, num_groups,
        group_size, rank, stream, thread_k, thread_n, sms, use_atomic_add, use_fp32_reduce, is_zp_float);
  }
}

template <typename T>
WorkspaceInfo get_workspace(bool use_fp32_reduce, bool has_act_order, int rank, int64_t size_m, int64_t size_k) {
  int sms = -1;
  hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, rank);

  size_t c_tmp = 0;  // empty
  if (use_fp32_reduce) {
    int max_m_block_size = (size_m + 16 - 1) / 16 * 16;
    max_m_block_size = min(max_m_block_size, 64);
    int max_c_tmp_size = sms * max_m_block_size * llm_kernels::nvidia::marlin::max_thread_n;
    c_tmp = max_c_tmp_size * sizeof(float);
  }

  size_t a_tmp = 0;  // empty
  if (has_act_order) {
    a_tmp = size_m * size_k * sizeof(T);
  }

  size_t workspace = 0;  // zeros
  int max_blocks_per_sm = 1;
  workspace = sms * max_blocks_per_sm;
  workspace = workspace * sizeof(int32_t);

  WorkspaceInfo info;
  info.c_tmp_size = c_tmp;
  info.a_tmp_size = a_tmp;
  info.workspace_size = workspace;

  return info;
}

template <typename T>
__global__ void permute_scales_kernel(const T* input, T* output, const size_t scale_perm_size, const size_t len) {
  size_t base = blockIdx.x * blockDim.x;
  size_t offset = threadIdx.x;

  int64_t scale_perm[64];
  int idx = 0;
  if (scale_perm_size == 64) {
    for (int i = 0; i < 8; i++) {
      for (int j = 0; j < 8; j++) {
        scale_perm[idx++] = i + 8 * j;
      }
    }
  } else {
    for (int i = 0; i < 4; i++) {
      for (int j : {0, 1, 8, 9, 16, 17, 24, 25}) {
        scale_perm[idx++] = 2 * i + j;
      }
    }
  }

  if (base + offset < len) {
    output[base + offset] = input[base + scale_perm[offset]];
  }
}

template <typename T>
void permute_scales(hipStream_t stream, const T* input, T* output, const size_t k, const size_t n,
                    const int64_t groupsize) {
  size_t scale_perm_size;
  if (groupsize < static_cast<int64_t>(k) && groupsize != -1) {
    scale_perm_size = 64;
  } else {
    scale_perm_size = 32;
  }
  KLLM_KERNEL_CHECK_WITH_INFO(scale_perm_size != 0, "scale_perm_size must not be 0");
  KLLM_KERNEL_CHECK_WITH_INFO(k % groupsize == 0, "k must can be divided by groupsize");
  size_t len = k / groupsize * n;
  KLLM_KERNEL_CHECK_WITH_INFO(len % scale_perm_size == 0, "k / groupsize * n must can be divided by scale_perm_size");
  dim3 blockSize(scale_perm_size);
  dim3 gridSize(len / scale_perm_size);
  permute_scales_kernel<T><<<gridSize, blockSize, 0, stream>>>(input, output, scale_perm_size, len);
}

template void gptq_marlin_gemm<float>(void* a, void* a_tmp, void* b_q_weight, void* b_scales, void* b_zeros,
                                      void* g_idx, void* perm, void* workspace, void* c, void* c_tmp, int64_t size_m,
                                      int64_t size_n, int64_t size_k, int64_t num_groups, bool is_k_full,
                                      bool use_atomic_add, bool use_fp32_reduce, bool is_zp_float, bool has_zp,
                                      bool has_act_order, bool is_awq, int rank, hipStream_t stream);
template void gptq_marlin_gemm<half>(void* a, void* a_tmp, void* b_q_weight, void* b_scales, void* b_zeros, void* g_idx,
                                     void* perm, void* workspace, void* c, void* c_tmp, int64_t size_m, int64_t size_n,
                                     int64_t size_k, int64_t num_groups, bool is_k_full, bool use_atomic_add,
                                     bool use_fp32_reduce, bool is_zp_float, bool has_zp, bool has_act_order,
                                     bool is_awq, int rank, hipStream_t stream);
template void gptq_marlin_gemm<__hip_bfloat16>(void* a, void* a_tmp, void* b_q_weight, void* b_scales, void* b_zeros,
                                              void* g_idx, void* perm, void* workspace, void* c, void* c_tmp,
                                              int64_t size_m, int64_t size_n, int64_t size_k, int64_t num_groups,
                                              bool is_k_full, bool use_atomic_add, bool use_fp32_reduce,
                                              bool is_zp_float, bool has_zp, bool has_act_order, bool is_awq, int rank,
                                              hipStream_t stream);

template WorkspaceInfo get_workspace<float>(bool use_fp32_reduce, bool has_act_order, int rank, int64_t size_m,
                                            int64_t size_k);
template WorkspaceInfo get_workspace<half>(bool use_fp32_reduce, bool has_act_order, int rank, int64_t size_m,
                                           int64_t size_k);
template WorkspaceInfo get_workspace<__hip_bfloat16>(bool use_fp32_reduce, bool has_act_order, int rank, int64_t size_m,
                                                    int64_t size_k);

template void permute_scales<float>(hipStream_t stream, const float* input, float* output, const size_t k,
                                    const size_t n, const int64_t groupsize);
template void permute_scales<half>(hipStream_t stream, const half* input, half* output, const size_t k, const size_t n,
                                   const int64_t groupsize);
template void permute_scales<__hip_bfloat16>(hipStream_t stream, const __hip_bfloat16* input, __hip_bfloat16* output,
                                            const size_t k, const size_t n, const int64_t groupsize);

}  // namespace marlin
}  // namespace nvidia
}  // namespace llm_kernels