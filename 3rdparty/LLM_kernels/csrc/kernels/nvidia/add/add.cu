#include "hip/hip_runtime.h"
/*
 * Copyright 2024 Tencent Inc.  All rights reserved.
 * Copyright (c) 2019-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "csrc/kernels/nvidia/add/add.h"

#include "csrc/utils/nvidia/cuda_bf16_fallbacks.cuh"
#include "csrc/utils/nvidia/cuda_type_utils.cuh"
#include "csrc/utils/nvidia/cuda_utils.h"

using namespace llm_kernels::utils;

namespace llm_kernels {
namespace nvidia {

constexpr int32_t ADD_BIAS_RES_BLOCK_SIZE = 1024;

template <typename T, int32_t RESIDUAL_NUM, typename T2 = T>
__global__ void AddBiasResidualKernel(T* output, const T2* __restrict__ input, const T* __restrict__ residual1,
                                      const T* __restrict__ residual2, const T* __restrict__ bias,
                                      const float* __restrict__ scale_inter, const float* __restrict__ scale_out,
                                      const int32_t total_element_num, const int32_t n) {
  const int32_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < total_element_num) {
    T bias_val = (bias == nullptr) ? (T)(0.0f) : bias[index % n];
    T in;
    if (std::is_same<T, T2>::value) {
      in = CastCudaDataType<T>(input[index]);  // cast required for compilation when T != T2
    } else {
      in = CastCudaDataType<float>(input[index]) * (*scale_inter) * (*scale_out);
    }

    if (RESIDUAL_NUM == 1) {
      if (residual1) {
        output[index] = in + residual1[index] + bias_val;
      } else {
        output[index] = in + bias_val;
      }
    } else if (RESIDUAL_NUM == 2) {
      output[index] = in + residual1[index] + residual2[index] + bias_val;
    }
  }
}

template <typename T>
__global__ void AddResidualKernel(T* output, const T* __restrict__ input, const T* __restrict__ residual,
                                  const int32_t total_element_num) {
  const int32_t index = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
  if (index < total_element_num) {
    const int32_t idx = index / 4;
    T input_val = input[idx];
    T residual_val = residual[idx];
    T output_val_tmp;

    output_val_tmp.x = input_val.x + residual_val.x;
    output_val_tmp.y = input_val.y + residual_val.y;
    output_val_tmp.z = input_val.z + residual_val.z;
    output_val_tmp.w = input_val.w + residual_val.w;
    output[idx] = output_val_tmp;
  }
}

template <typename T>
void InvokeAddBiasResidual(T* output, const T* input, const T* residual1, const T* residual2, const T* bias,
                           const float* scale_inter, const float* scale_out, const int32_t m, const int32_t n,
                           hipStream_t stream) {
  if (((scale_inter == nullptr) ^ (scale_out == nullptr))) {
    throw std::runtime_error("Cannot use `scale_inter` without `scale_out`");
  }
  constexpr int32_t BLOCK_SIZE = 256;
  int32_t total_element_num = m * n;
  const bool should_scale_input = scale_inter != nullptr;
  if (should_scale_input) {
    int32_t block_num = ceil(float(m * n) / BLOCK_SIZE);
    dim3 grid(block_num);
    dim3 block(BLOCK_SIZE);
    if (residual2 == nullptr) {
      AddBiasResidualKernel<T, 1><<<grid, block, 0, stream>>>(output, reinterpret_cast<const int32_t*>(input),
                                                              residual1, residual2, bias, scale_inter, scale_out,
                                                              total_element_num, n);
    } else {
      AddBiasResidualKernel<T, 2><<<grid, block, 0, stream>>>(output, reinterpret_cast<const int32_t*>(input),
                                                              residual1, residual2, bias, scale_inter, scale_out,
                                                              total_element_num, n);
    }
  } else {
    const size_t kVecSize = 4;
    int32_t block_num = ceil(float(total_element_num) / BLOCK_SIZE);
    if (bias == nullptr && residual2 == nullptr && total_element_num % kVecSize == 0) {
      block_num = ceil(float(block_num) / kVecSize);
      dim3 grid(block_num);
      dim3 block(BLOCK_SIZE);
      using VecType = typename utils::PackType<T, kVecSize>::type;
      AddResidualKernel<VecType>
          <<<grid, block, 0, stream>>>(reinterpret_cast<VecType*>(output), reinterpret_cast<const VecType*>(input),
                                       reinterpret_cast<const VecType*>(residual1), total_element_num);
    } else {
      dim3 grid(block_num);
      dim3 block(BLOCK_SIZE);
      if (residual2 == nullptr) {
        AddBiasResidualKernel<T, 1><<<grid, block, 0, stream>>>(output, input, residual1, residual2, bias, nullptr,
                                                                nullptr, total_element_num, n);
      } else {
        AddBiasResidualKernel<T, 2><<<grid, block, 0, stream>>>(output, input, residual1, residual2, bias, nullptr,
                                                                nullptr, total_element_num, n);
      }
    }
  }
}

template <typename T>
void InvokeAddBiasResidual(T* output, const T* residual1, const T* residual2, const T* bias, const int32_t m,
                           const int32_t n, hipStream_t stream) {
  InvokeAddBiasResidual(output, output, residual1, residual2, bias, nullptr, nullptr, m, n, stream);
}

template <typename T>
__global__ void AddBiasAttentionFfnResidualKernel(T* block_output, const T* __restrict__ ffn_output,
                                                  const T* __restrict__ attn_output, const T* __restrict__ block_input,
                                                  const T* __restrict__ bias, const int32_t m, const int32_t n,
                                                  const int32_t block_input_tp_split) {
  const int32_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  if (col_index < n) {
    block_output[blockIdx.x * n + col_index] =
        ffn_output[blockIdx.x * n + col_index] + attn_output[blockIdx.x * n + col_index] + bias[col_index] +
        ((block_input != nullptr)
             ? CastCudaDataType<T>((float)block_input[blockIdx.x * n + col_index] / (float)block_input_tp_split)
             : static_cast<T>(0.0f));
  }
}

template <typename T>
__global__ void AddBiasAttentionFfnResidualKernel(T* block_output, const T* __restrict__ ffn_output,
                                                  const T* __restrict__ attn_output, const T* __restrict__ bias,
                                                  const int32_t m, const int32_t n,
                                                  const int32_t block_input_tp_split) {
  const int32_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  if (col_index < n) {
    const int32_t global_index = blockIdx.x * n + col_index;
    block_output[global_index] =
        add(CastCudaDataType<T>((float)block_output[global_index] / (float)block_input_tp_split),
            ffn_output[global_index], attn_output[global_index], bias[col_index]);
  }
}

template <typename T>
void InvokeAddBiasAttentionFfnResidual(T* block_output, const T* ffn_output, const T* attn_output, const T* block_input,
                                       const T* bias, const int32_t m, const int32_t n,
                                       const int32_t block_input_tp_split, hipStream_t stream) {
  int32_t blocks_per_row = ceil(float(n) / ADD_BIAS_RES_BLOCK_SIZE);
  dim3 grid(m, blocks_per_row);
  dim3 block(min(n, ADD_BIAS_RES_BLOCK_SIZE));
  if (block_output == block_input) {
    AddBiasAttentionFfnResidualKernel<<<grid, block, 0, stream>>>(block_output, ffn_output, attn_output, bias, m, n,
                                                                  block_input_tp_split);
  } else {
    AddBiasAttentionFfnResidualKernel<<<grid, block, 0, stream>>>(block_output, ffn_output, attn_output, block_input,
                                                                  bias, m, n, block_input_tp_split);
  }
}

#define INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL(T)                                                          \
  template void InvokeAddBiasResidual(T* output, const T* input, const T* residual1, const T* residual2, \
                                      const T* bias, const float* scale_inter, const float* scale_out,   \
                                      const int32_t m, const int32_t n, hipStream_t stream)
INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL(float);
INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL(half);
INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL(__hip_bfloat16);
#undef INSTANTIATE_INVOKE_ADD_BIAS_RESIDUAL

template void InvokeAddBiasResidual(float* output, const float* residual1, const float* residual2, const float* bias,
                                    const int32_t m, const int32_t n, hipStream_t stream);

template void InvokeAddBiasResidual(half* output, const half* residual1, const half* residual2, const half* bias,
                                    const int32_t m, const int32_t n, hipStream_t stream);
template void InvokeAddBiasResidual(__hip_bfloat16* output, const __hip_bfloat16* residual1,
                                    const __hip_bfloat16* residual2, const __hip_bfloat16* bias, const int32_t m,
                                    const int32_t n, hipStream_t stream);

template void InvokeAddBiasAttentionFfnResidual(float* block_output, const float* ffn_output, const float* attn_output,
                                                const float* input, const float* bias, const int32_t m, const int32_t n,
                                                const int32_t block_input_tp_split, hipStream_t stream);

template void InvokeAddBiasAttentionFfnResidual(half* block_output, const half* ffn_output, const half* attn_output,
                                                const half* input, const half* bias, const int32_t m, const int32_t n,
                                                const int32_t block_input_tp_split, hipStream_t stream);
template void InvokeAddBiasAttentionFfnResidual(__hip_bfloat16* block_output, const __hip_bfloat16* ffn_output,
                                                const __hip_bfloat16* attn_output, const __hip_bfloat16* input,
                                                const __hip_bfloat16* bias, const int32_t m, const int32_t n,
                                                const int32_t block_input_tp_split, hipStream_t stream);

// NOTE(karlluo): inplace case output can't use __restrict__
template <typename T>
__global__ void InvokeT5addResidualKernel(T* output, const T* __restrict__ input, const int32_t m, const int32_t n) {
  const int32_t col_index = blockIdx.y * blockDim.x + threadIdx.x;
  if (col_index < n) {
    float out_val = (float)output[blockIdx.x * n + col_index] + (float)input[blockIdx.x * n + col_index];
    output[blockIdx.x * n + col_index] =
        (T)((std::is_same<T, half>::value && (out_val > 64512 || out_val < -64512)) ? (out_val > 0 ? 64512 : -64512)
                                                                                    : out_val);
  }
}

template <typename T>
void InvokeT5AddResidual(T* output, const T* input, const int32_t m, const int32_t n, hipStream_t stream) {
  int32_t blocks_per_row = ceil(float(n) / ADD_BIAS_RES_BLOCK_SIZE);
  dim3 grid(m, blocks_per_row);
  dim3 block(min(n, ADD_BIAS_RES_BLOCK_SIZE));
  InvokeT5addResidualKernel<<<grid, block, 0, stream>>>(output, input, m, n);
}

template void InvokeT5AddResidual(float* output, const float* input, const int32_t m, const int32_t n,
                                  hipStream_t stream);
template void InvokeT5AddResidual(half* output, const half* input, const int32_t m, const int32_t n,
                                  hipStream_t stream);
template void InvokeT5AddResidual(__hip_bfloat16* output, const __hip_bfloat16* input, const int32_t m, const int32_t n,
                                  hipStream_t stream);

template <typename T>
void InvokeT5AddBiasResidual(T* output, const T* input, const T* bias, const int32_t m, const int32_t n,
                             hipStream_t stream) {
  if (bias != nullptr) {
    InvokeAddBiasResidual(output, input, bias, m, n, stream);
  } else {
    InvokeT5AddResidual(output, input, m, n, stream);
  }
  return;
}

template void InvokeT5AddBiasResidual(float* output, const float* input, const float* bias, const int32_t m,
                                      const int32_t n, hipStream_t stream);
template void InvokeT5AddBiasResidual(half* output, const half* input, const half* bias, const int32_t m,
                                      const int32_t n, hipStream_t stream);
template void InvokeT5AddBiasResidual(__hip_bfloat16* output, const __hip_bfloat16* input, const __hip_bfloat16* bias,
                                      const int32_t m, const int32_t n, hipStream_t stream);

// InvokeAddBiasResidualCol32 input1/input2/out matrix with layout of cublasLt CUBLASLT_ORDER_COL32 (m*n) (grid, block)
// must be (m, n/4) using char4
template <typename T>
__global__ void AddBiasInputCOL32Int8IDataTypeOKernel(T* output, const int8_t* __restrict__ input1,
                                                      const T* __restrict__ input2, const T* __restrict__ bias,
                                                      int32_t m, int32_t n,
                                                      const float* __restrict__ input1_deq_factor_ptr) {
  const float input1_deQFactor = __ldg(input1_deq_factor_ptr);
  int32_t col_start = threadIdx.x << 2;

  float local_out[4];
  int32_t outIdx = ((col_start & 0xffffffe0) * m + (blockIdx.x << 5) + (col_start & 31)) >> 2;
  char4* input1TmpPtr = (char4*)input1;
  char4 input1Tmp = __ldg(input1TmpPtr + outIdx);

  int32_t col_start_tmp = col_start;
  local_out[0] = static_cast<float>(input2[(outIdx << 2) + 0]) + static_cast<float>(input1Tmp.x) * input1_deQFactor +
                 static_cast<float>(__ldg(bias + col_start_tmp));
  col_start_tmp = col_start_tmp + 1;
  local_out[1] = static_cast<float>(input2[(outIdx << 2) + 1]) + static_cast<float>(input1Tmp.y) * input1_deQFactor +
                 static_cast<float>(__ldg(bias + col_start_tmp));
  col_start_tmp = col_start_tmp + 1;
  local_out[2] = static_cast<float>(input2[(outIdx << 2) + 2]) + static_cast<float>(input1Tmp.z) * input1_deQFactor +
                 static_cast<float>(__ldg(bias + col_start_tmp));
  col_start_tmp = col_start_tmp + 1;
  local_out[3] = static_cast<float>(input2[(outIdx << 2) + 3]) + static_cast<float>(input1Tmp.w) * input1_deQFactor +
                 static_cast<float>(__ldg(bias + col_start_tmp));

  for (int32_t i = 0; i < 4; i++) {
    output[(outIdx << 2) + i] = static_cast<T>(local_out[i]);
  }
}

template <>
__global__ void AddBiasInputCOL32Int8IDataTypeOKernel(half4* output, const int8_t* __restrict__ input1,
                                                      const half4* __restrict__ input2, const half4* __restrict__ bias,
                                                      int32_t m, int32_t n,
                                                      const float* __restrict__ input1_deq_factor_ptr) {
  const float input1_deQFactor = __ldg(input1_deq_factor_ptr);
  int32_t col_start = (blockIdx.x << 5) + (threadIdx.x << 2);
  int32_t row_start = (blockIdx.y << 5) + (threadIdx.y);

  if (col_start < n && row_start < m) {
    half4 local_out;
    int32_t outIdx = ((col_start & 0xffffffe0) * m + (row_start << 5) + (col_start & 31)) >> 2;
    char4* input1TmpPtr = (char4*)input1;
    char4 input1Tmp = input1TmpPtr[outIdx];
    half4 input2Tmp = input2[outIdx];
    half4 biasTmp = bias[col_start >> 2];

    local_out.x = static_cast<half>((float)input1Tmp.x * input1_deQFactor + (float)biasTmp.x + (float)input2Tmp.x);
    local_out.y = static_cast<half>((float)input1Tmp.y * input1_deQFactor + (float)biasTmp.y + (float)input2Tmp.y);
    local_out.z = static_cast<half>((float)input1Tmp.z * input1_deQFactor + (float)biasTmp.z + (float)input2Tmp.z);
    local_out.w = static_cast<half>((float)input1Tmp.w * input1_deQFactor + (float)biasTmp.w + (float)input2Tmp.w);
    output[outIdx] = local_out;
  }
}

template <typename T>
void InvokeAddBiasResidualCol32(T* output, const int8_t* input1, const T* input2, const T* bias, int32_t m, int32_t n,
                                hipStream_t stream, const float* input1_deq_factor_ptr) {
  dim3 grid((n + DEFAULT_CUDA_WARP_SIZE - 1) / DEFAULT_CUDA_WARP_SIZE,
            (m + DEFAULT_CUDA_WARP_SIZE - 1) / DEFAULT_CUDA_WARP_SIZE);
  dim3 block(DEFAULT_CUDA_QUARTER_WARP_SIZE, DEFAULT_CUDA_WARP_SIZE);

  if (sizeof(T) == 2) {
    AddBiasInputCOL32Int8IDataTypeOKernel<<<grid, block, 0, stream>>>((half4*)output, input1, (const half4*)input2,
                                                                      (const half4*)bias, m, n, input1_deq_factor_ptr);
  } else {
    AddBiasInputCOL32Int8IDataTypeOKernel<T>
        <<<grid, block, 0, stream>>>(output, input1, input2, bias, m, n, input1_deq_factor_ptr);
  }
}

template void InvokeAddBiasResidualCol32(float* output, const int8_t* input1, const float* input2, const float* bias,
                                         int32_t m, int32_t n, hipStream_t stream, const float* input1_deq_factor_ptr);

template void InvokeAddBiasResidualCol32(half* output, const int8_t* input1, const half* input2, const half* bias,
                                         int32_t m, int32_t n, hipStream_t stream, const float* input1_deq_factor_ptr);

// InvokeAddBiasResidualCol32 input1/input2/out matrix with layout of cublasLt CUBLASLT_ORDER_COL32 (m*n) (grid, block)
// must be (m, n/4) using char4
template <typename T>
__global__ void AddBiasInputCOL32Int32IDataTypeOKernel(T* output, const int32_t* __restrict__ input1,
                                                       const T* __restrict__ input2, const T* __restrict__ bias,
                                                       int32_t m, int32_t n, const float* __restrict__ weight_amax,
                                                       const float* __restrict__ input1_amax_ptr,
                                                       const int32_t scale_is_vector) {
  int32_t col_start = threadIdx.x << 2;
  const float4* weight_scale_ptr = (const float4*)weight_amax;
  const float4 weight_scale = __ldg(weight_scale_ptr + threadIdx.x * scale_is_vector);
  const float input1_deQ = __ldg(input1_amax_ptr) / 127.0f;

  float local_out[4];
  int32_t outIdx = ((col_start & 0xffffffe0) * m + (blockIdx.x << 5) + (col_start & 31)) >> 2;
  int4* input1TmpPtr = (int4*)input1;
  int4 input1Tmp = input1TmpPtr[outIdx];

  int32_t col_start_tmp = col_start;
  local_out[0] = static_cast<float>(input2[(outIdx << 2) + 0]) +
                 static_cast<float>(input1Tmp.x) * input1_deQ * weight_scale.x / 127.0f +
                 static_cast<float>(__ldg(bias + col_start_tmp));
  col_start_tmp = col_start_tmp + 1;
  local_out[1] = static_cast<float>(input2[(outIdx << 2) + 1]) +
                 static_cast<float>(input1Tmp.y) * input1_deQ * weight_scale.y / 127.0f +
                 static_cast<float>(__ldg(bias + col_start_tmp));
  col_start_tmp = col_start_tmp + 1;
  local_out[2] = static_cast<float>(input2[(outIdx << 2) + 2]) +
                 static_cast<float>(input1Tmp.z) * input1_deQ * weight_scale.z / 127.0f +
                 static_cast<float>(__ldg(bias + col_start_tmp));
  col_start_tmp = col_start_tmp + 1;
  local_out[3] = static_cast<float>(input2[(outIdx << 2) + 3]) +
                 static_cast<float>(input1Tmp.w) * input1_deQ * weight_scale.w / 127.0f +
                 static_cast<float>(__ldg(bias + col_start_tmp));

  for (int32_t i = 0; i < 4; i++) {
    output[(outIdx << 2) + i] = static_cast<T>(local_out[i]);
  }
}

template <>
__global__ void AddBiasInputCOL32Int32IDataTypeOKernel(half4* output, const int32_t* __restrict__ input1,
                                                       const half4* __restrict__ input2, const half4* __restrict__ bias,
                                                       int32_t m, int32_t n, const float* __restrict__ weight_amax,
                                                       const float* __restrict__ input1_amax_ptr,
                                                       const int32_t scale_is_vector) {
  int32_t col_start = threadIdx.x << 2;
  const float4* weight_scale_ptr = (const float4*)weight_amax;
  const float weight_scale_single = __ldg(weight_amax);
  const float4 weight_scale = scale_is_vector == 1 ? __ldg(weight_scale_ptr + threadIdx.x * scale_is_vector)
                                                   : make_float4(weight_scale_single, weight_scale_single,
                                                                 weight_scale_single, weight_scale_single);
  const float input1_deQ = __ldg(input1_amax_ptr) / 127.0f;

  float local_out[4];
  int32_t outIdx = ((col_start & 0xffffffe0) * m + (blockIdx.x << 5) + (col_start & 31)) >> 2;
  int4* input1TmpPtr = (int4*)input1;
  int4 input1Tmp = input1TmpPtr[outIdx];
  half4 input2Tmp = input2[outIdx];
  half4 biasTmp = bias[threadIdx.x];

  local_out[0] = static_cast<float>(input2Tmp.x) +
                 static_cast<float>(input1Tmp.x) * input1_deQ * weight_scale.x / 127.0f + static_cast<float>(biasTmp.x);
  local_out[1] = static_cast<float>(input2Tmp.y) +
                 static_cast<float>(input1Tmp.y) * input1_deQ * weight_scale.y / 127.0f + static_cast<float>(biasTmp.y);
  local_out[2] = static_cast<float>(input2Tmp.z) +
                 static_cast<float>(input1Tmp.z) * input1_deQ * weight_scale.z / 127.0f + static_cast<float>(biasTmp.z);
  local_out[3] = static_cast<float>(input2Tmp.w) +
                 static_cast<float>(input1Tmp.w) * input1_deQ * weight_scale.w / 127.0f + static_cast<float>(biasTmp.w);

  half4 outTmp;
  outTmp.x = static_cast<half>(local_out[0]);
  outTmp.y = static_cast<half>(local_out[1]);
  outTmp.z = static_cast<half>(local_out[2]);
  outTmp.w = static_cast<half>(local_out[3]);

  output[outIdx] = outTmp;
}

template <typename T>
void InvokeAddBiasResidualCol32(T* output, const int32_t* input1, const T* input2, const T* bias, int32_t m, int32_t n,
                                hipStream_t stream, const float* weight_amax, const float* input1_amax_ptr,
                                const int32_t scale_is_vector) {
  dim3 grid(m);
  dim3 block(n >> 2);
  if (block.x > ADD_BIAS_RES_BLOCK_SIZE) {
    throw std::runtime_error("block dim x is bigger than 1024");
  }

  if (sizeof(T) == 2) {
    AddBiasInputCOL32Int32IDataTypeOKernel<<<grid, block, 0, stream>>>((half4*)output, input1, (const half4*)input2,
                                                                       (const half4*)bias, m, n, weight_amax,
                                                                       input1_amax_ptr, scale_is_vector);
  } else {
    AddBiasInputCOL32Int32IDataTypeOKernel<T>
        <<<grid, block, 0, stream>>>(output, input1, input2, bias, m, n, weight_amax, input1_amax_ptr, scale_is_vector);
  }
}

template void InvokeAddBiasResidualCol32(float* output, const int32_t* input1, const float* input2, const float* bias,
                                         int32_t m, int32_t n, hipStream_t stream, const float* weight_amax,
                                         const float* input1_amax_ptr, const int32_t scale_is_vector);

template void InvokeAddBiasResidualCol32(half* output, const int32_t* input1, const half* input2, const half* bias,
                                         int32_t m, int32_t n, hipStream_t stream, const float* weight_amax,
                                         const float* input1_amax_ptr, const int32_t scale_is_vector);

}  // namespace nvidia
}  // namespace llm_kernels
